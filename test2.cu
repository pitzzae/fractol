
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA(int **f)
{
  printf("Hello thread %d, %d, %d, %d, %d, %d, f=%d\n", 
  	threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, f[threadIdx.y][threadIdx.x]);
}

int main()
{
	int	H = 32;
	int	W = 32;

	int *h_A = new int[W];
	for (int i = 0; i < W; i++)
	{
	    h_A[i] = i;
	    printf("%d\n", i);
	}

	int	**h_AA;
	h_AA = (int**)malloc(H* sizeof(int*));
	for (int i = 0; i < H; i++)
	{
	    hipMalloc((void **)&h_AA[i], W * sizeof(int));
	    hipMemcpy(h_AA[i], h_A, W * sizeof(int), hipMemcpyHostToDevice);
	}

	int	**d_AA;
	hipMalloc(&d_AA, H * sizeof(int*));
	hipMemcpy(d_AA, h_AA, H * sizeof(int*), hipMemcpyHostToDevice);

	dim3 dimBlock(H, W);
	dim3 dimGrid(100, 100);
	helloCUDA<<<dimGrid, dimBlock>>>(d_AA);
	printf("%s\n", "END");
	hipDeviceReset();
	return 0;
}
//nvcc -arch=sm_20 test.cu -run