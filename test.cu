// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 7;
const int blocksize = 7;

__device__
int		ft_strcmp(const char *s1, const char *s2)
{
	int count;

	count = 0;
	while (s1[count] == s2[count] && s1[count] != '\0')
		count++;
	return (((unsigned char)s1[count] - (unsigned char)s2[count]));
}

__device__
char *test(void)
{
	return ("NOK");
}

__global__
void hello(char *a, int *b)
{
	if (ft_strcmp("OK", test()) == 0)
		a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	char a[N] = "Hello ";
	int b[N] = {15, 10, 6, 0, -11, 1, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	//printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );

	printf("%s\n", a);
	return EXIT_SUCCESS;
}
