#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   fractal.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gtorresa <gtorresa@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/02/13 16:44:36 by gtorresa          #+#    #+#             */
/*   Updated: 2016/02/19 15:26:41 by gtorresa         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "fractol.h"
}

#include <stdio.h>

__device__
t_complex	ft_cplx_add_cuda(t_complex z1, t_complex z2)
{
	t_complex		r;

	r.real = z1.real + z2.real;
	r.imag = z1.imag + z2.imag;
	return (r);
}

__device__
t_complex	ft_cplx_mul_cuda(t_complex z1, t_complex z2)
{
	t_complex		r;

	r.real = z1.real * z2.real - z1.imag * z2.imag;
	r.imag = z1.real * z2.imag + z1.imag * z2.real;
	return (r);
}

__device__
t_complex	ft_cplx_pow_cuda(t_complex z1, int p)
{
	t_complex		r;
	int				loop;

	loop = 0;
	r.real = z1.real;
	r.imag = z1.imag;
	if (p > 0)
	{
		while (loop < p)
		{
			r = ft_cplx_mul_cuda(r, z1);
			loop++;
		}
	}
	return (r);
}

__global__
void ft_cplx_cuda(double *i, double *j)
{
	t_complex		z[2];

	z[0].real = i[0];
	z[0].imag = i[1];
	z[1].real = j[0];
	z[1].imag = j[1];
	z[1] = ft_cplx_add_cuda(ft_cplx_mul_cuda(z[1], z[1]), z[0]);
	j[0] = z[1].real;
	j[1] = z[1].imag;
}

double				*convert_cplx(t_complex z)
{
	double			*c;

	c = (double*)malloc(2*sizeof(double));
	c[0] = z.real;
	c[1] = z.imag;
	return(c);
}

void				fractal_julia(t_complex z[2])
{
	double			*i;
	double			*j;
	double			*res;
	int				N;
	int 			size;

	double *test1;
	double *test2;

	test1 = convert_cplx(z[0]);
	test2 = convert_cplx(z[1]);

	N = 2;
	size = N*sizeof(double);
	hipMalloc((void**)&i, size);
	hipMalloc((void**)&j, size);
	res = (double*)malloc(2*sizeof(double));

	hipMemcpy(i, test1, size, hipMemcpyHostToDevice);
	hipMemcpy(j, test2, size, hipMemcpyHostToDevice);

	dim3 dimBlock(1, 1);
	dim3 dimGrid(1, 1);
	ft_cplx_cuda<<<dimGrid, dimBlock>>>(i, j);

	hipMemcpy(res, j, size, hipMemcpyDeviceToHost);
	hipFree(i);
	hipFree(j);
	free(test1);
	free(test2);
}

void				fractal_julia1(t_complex z[2])
{
	z[1] = ft_cplx_add(ft_cplx_pow(z[1], 3), z[0]);
}

void				fractal_mandelbrot(t_complex z[2])
{
	z[1] = ft_cplx_add(ft_cplx_pow(z[1], 3), z[1]);
}

void				fractal_mandelbrot1(t_complex z[2])
{
	z[1] = ft_cplx_add(ft_cplx_pow(z[1], 5), z[1]);
}

void				fractal_mandelbrot2(t_complex z[2])
{
	z[1] = ft_cplx_add(ft_cplx_pow(z[1], 8), z[1]);
}
