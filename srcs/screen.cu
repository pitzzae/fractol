#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   screen.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gtorresa <gtorresa@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/02/13 16:13:28 by gtorresa          #+#    #+#             */
/*   Updated: 2016/02/19 13:44:46 by gtorresa         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "fractol.h"
}

static void		put_pixel_to_img(double px[2], t_data *e, int color)
{
	int i;

	i = ((int)px[0] * 4) + ((int)px[1] * e->img_line);
	e->pixel_img[i] = color;
	e->pixel_img[++i] = color >> 8;
	e->pixel_img[++i] = color >> 16;
}

#include <stdio.h>

static int		get_color_px(t_complex z[2], int max_c)
{
	int				c;
	int				d;
	int				g;
	int				test = 0;

	d = 0;
	g = 0;
	c = 0x000000;
	while (ft_cabs(z[1]) < 4 && c < max_c)
	{
		//fract(z);
		if (d < 50)
		{
			c += 0x00800;
			d = 0;
		}
		if ((g % 3) == 0)
			c += 0x060000;
		c += 0x000015;
		d++;
		g++;
		test++;
	}
	//printf("%d\n", test);
	return (c);
}

static void		ft_put_pixel(t_data *data, int px[2])
{
	t_complex		z[2];
	double			px2[2];
	int				color;

	z[0].real = data->c_point.real;
	z[0].imag = data->c_point.imag;
	z[1].real = ((px[0] / data->zoom) + data->ref_x);
	z[1].imag = ((px[1] / data->zoom + data->ref_y));
	px2[0] = px[0] + (W_W / 2);
	px2[1] = px[1] + (W_H / 2);
	color = get_color_px(z, 0xFFFFFF);
	put_pixel_to_img(px2, data, color);
}

__global__
void hello(int **in)
{
	in[threadIdx.x][threadIdx.y]++;
}

void			*print_screen(void *s_data)
{
	double		px[2];
	t_thread	*data;
	int			*s_c;

	//hipMalloc((void**)&s_c, W_H * W_W * sizeof(int));
	data = (t_thread*)s_data;
	px[0] = data->s.start;
	printf("%d\n", data->s.start);
	printf("%d\n", data->s.len);
	/*while (px[0] < data->s.len)
	{
		px[1] = -(W_H / 2);
		printf("%s\n", "test1");
		while (px[1] < W_H / 2)
		{
			printf("%s\n", "test2");
			put_pixel_to_img(px, data->d, 127);
			printf("%s\n", "test5");
			px[1]++;
		}
		px[0]++;
	}*/

	int			**line;
	int			pos_line;
	int			init_val;

	pos_line = 0;
	init_val = 0;
	hipMalloc((void**)&line, W_H * sizeof(*int));
	while (pos_line < W_W)
	{
		hipMalloc((void**)&line[pos_line], W_W * sizeof(*int));
		hipMemcpy(line[pos_line] , init_val, W_W * sizeof(*int), hipMemcpyHostToDevice);
		pos_line++;
	}

	hipDeviceReset();
	pthread_exit(NULL);
}
